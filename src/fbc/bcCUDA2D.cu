#include "hip/hip_runtime.h"
//===============================================================================
// This file is part of the software Fast Box-Counting:
// https://www.ugr.es/~demiras/fbc
//
// Copyright(c)2022 University of Granada - SPAIN
//
// FOR RESEARCH PURPOSES ONLY.THE SOFTWARE IS PROVIDED "AS IS," AND THE
// UNIVERSITY OF GRANADA DO NOT MAKE ANY WARRANTY, EXPRESS OR IMPLIED, INCLUDING
// BUT NOT LIMITED TO WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE, NOR DO THEY ASSUME ANY LIABILITY OR RESPONSIBILITY FOR THE USE
// OF THIS SOFTWARE.
//
// For more information see license.html
// ===============================================================================
//
// Authors: Juan Ruiz de Miras and Miguel �ngel Posadas, 2022
// Contact: demiras@ugr.es

#include <iostream>
#include <iomanip>
#include <stdio.h>

#include "bcCUDA2D.cuh"

__global__ void BCKernel2D(unsigned char* M, const long long int m, const unsigned long long int bits_m, const unsigned long long int sm,
						   const unsigned long long bits_s, const unsigned long long bits_TPB, unsigned int* n)
{
	register unsigned long long int tid = threadIdx.x; 
	register unsigned long long int idx = (blockIdx.x << bits_TPB) + tid; // 2 ^ bits_TPB = TPB

	// identifies grid index (i, j) from block and thread values
	register unsigned long long int i = idx >> (bits_m - bits_s); // i index: idx / (m/s)
	register unsigned long long int j = idx & ((m >> bits_s) - 1); // j index: idx mod (m/s)

	// global location of position (0, 0) of the grid (i, j) is gi + gj
	const register unsigned long long int gi = (i << bits_s) << bits_m;  
	const register unsigned long long int gism = ((i << bits_s) + sm) << bits_m;  
	const register unsigned long long int gj = j << bits_s;  
	const register unsigned long long int gjsm = (j << bits_s) +sm;  

	// compute and store the occupancy value of the grid (i, j)
	M[gi + gj] = M[gi + gj] || M[gi + gjsm] || M[gism + gj] || M[gism + gjsm];
	atomicAdd(n, M[gi + gj]);
}

hipError_t CudaBC2D(const unsigned char* M, const long long int m, const unsigned char bits_m, const unsigned int TPB, const int nn, unsigned int* n)
{
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return cudaStatus;
    }

	// CPU-GPU data transfers
	unsigned char* device_M = 0; // matrix M in GPU device
	unsigned int* device_n = 0; // array n in GPU device
	cudaStatus = hipMalloc((void**)&device_M, m * m * sizeof(unsigned char));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
		return cudaStatus;
	}
    cudaStatus = hipMalloc((void**)&device_n, nn * sizeof(unsigned int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
		hipFree(device_M);
		return cudaStatus;
	}
	cudaStatus = hipMemcpy(device_M, M, m * m * sizeof(unsigned char), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		hipFree(device_M);
		hipFree(device_n);
		return cudaStatus;
	}
	cudaStatus = hipMemset(device_n, 0, nn * sizeof(unsigned int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		hipFree(device_M);
		hipFree(device_n);
		return cudaStatus;
	}

	dim3 grid, block(TPB, 1); // variables for kernel launching
	unsigned int s = 2;
	unsigned int size = m;
	unsigned char ni = 0;
	unsigned char bits_TPB = log(TPB)/log(2); // 2^bits_TPB = TPB

	while (size > 2) {
		unsigned int sm = s >> 1; // sm = s/2
		unsigned long long int num_box = (m*m) / (s*s);
			
		if (num_box >= TPB) {
			grid.x = ceilf(num_box / (float)TPB); // m/s *m/s = (grid_size * TPB)
			grid.y = 1;
		} else {
			grid.x = 1; grid.y = 1;
			block.x = num_box; block.y = 1;
			bits_TPB = log(num_box)/log(2);
		}

		// BCKernel call. Compute box-counting for grids of size s x s
		BCKernel2D<<<grid, block>>>(device_M, m, bits_m, sm, ni + 1, bits_TPB, &device_n[ni]);

		ni++;
		s <<= 1;
		size >>= 1;
	}
	//hipDeviceSynchronize();

	// GPU-CPU data transfer of the box-counting results
	cudaStatus = hipMemcpy(n, device_n, nn * sizeof(unsigned int), hipMemcpyDeviceToHost);

	hipFree(device_M);
	hipFree(device_n);
	return cudaStatus;
}

