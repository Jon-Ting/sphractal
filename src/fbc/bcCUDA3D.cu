#include "hip/hip_runtime.h"
//===============================================================================
// This file is part of the software Fast Box-Counting:
// https://www.ugr.es/~demiras/fbc
//
// Copyright(c)2022 University of Granada - SPAIN
//
// FOR RESEARCH PURPOSES ONLY.THE SOFTWARE IS PROVIDED "AS IS," AND THE
// UNIVERSITY OF GRANADA DO NOT MAKE ANY WARRANTY, EXPRESS OR IMPLIED, INCLUDING
// BUT NOT LIMITED TO WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE, NOR DO THEY ASSUME ANY LIABILITY OR RESPONSIBILITY FOR THE USE
// OF THIS SOFTWARE.
//
// For more information see license.html
// ===============================================================================
//
// Authors: Juan Ruiz de Miras and Miguel �ngel Posadas, 2022
// Contact: demiras@ugr.es

#include <iostream>
#include <iomanip>
#include <stdio.h>

#include "bcCUDA3D.cuh"

__global__ void BCKernel3D(unsigned char* M, const long long int m, const unsigned long long int bits_m, const unsigned long long int sm,
						   const unsigned long long bits_s, const unsigned long long bits_TPB, unsigned int* n)
{
	register unsigned long long int tid = threadIdx.x;
	register unsigned long long int idx = (blockIdx.x << bits_TPB) + tid; // 2 ^ bits_TPB = TPB

	// identifies grid index (i, j, k) from block and thread values
	register unsigned int k = idx >> ((bits_m - bits_s) + (bits_m - bits_s)); // k index: idx / ((m/s)*(m/s))
	register unsigned int offset = (idx & (((m >> bits_s) << (bits_m - bits_s)) - 1)); // (idx mod ((m/s)*(m/s))), offset inside k slice

	register unsigned long long int i = offset >> (bits_m - bits_s); // i index: offset / (m/s)
	register unsigned long long int j = offset & ((m >> bits_s) - 1); // j index: offset mod (m/s)

	// global location of position (0, 0, 0) of the grid (i, j, k) is gk + gi + gj
	const register unsigned long long int gi = (i << bits_s) << bits_m;
	const register unsigned long long int gj = j << bits_s;
	const register unsigned long long int gk = ((k << bits_s) << bits_m) << bits_m;

	const register unsigned long long int gism = ((i << bits_s) + sm) << bits_m;
	const register unsigned long long int gjsm = (j << bits_s) + sm;
	const register unsigned long long int gksm = (((k << bits_s) + sm) << bits_m) << bits_m;

	// compute and store the occupancy value of the grid (i, j, k)
	M[gk + gi + gj] = M[gk + gi + gj] || M[gk + gi + gjsm] || M[gk + gism + gj] || M[gk + gism + gjsm] || 
					  M[gksm + gi + gj] || M[gksm + gi + gjsm] || M[gksm + gism + gj] || M[gksm + gism + gjsm];
	atomicAdd(n, M[gk + gi + gj]);
}

hipError_t CudaBC3D(const unsigned char* M, const long long int m, const unsigned char bits_m, const unsigned int TPB, const int nn, unsigned int* n)
{
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return cudaStatus;
	}

	// CPU-GPU data transfers
	unsigned char* device_M = 0; // matrix M in GPU device
	unsigned int* device_n = 0; // array n in GPU device
	cudaStatus = hipMalloc((void**)&device_M, m * m * m * sizeof(unsigned char));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		return cudaStatus;
	}
	cudaStatus = hipMalloc((void**)&device_n, nn * sizeof(unsigned int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		hipFree(device_M);
		return cudaStatus;
	}
	cudaStatus = hipMemcpy(device_M, M, m * m * m * sizeof(unsigned char), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		hipFree(device_M);
		hipFree(device_n);
		return cudaStatus;
	}
	cudaStatus = hipMemset(device_n, 0, nn * sizeof(unsigned int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		hipFree(device_M);
		hipFree(device_n);
		return cudaStatus;
	}

	dim3 grid, block(TPB, 1, 1); // variables for kernel launching
	unsigned int s = 2;
	unsigned int size = m;
	unsigned char ni = 0;
	unsigned char bits_TPB = log(TPB) / log(2); // 2^bits_TPB = TPB

	while (size > 2) {
		unsigned int sm = s >> 1; // sm = s/2
		unsigned long long int num_box = (m * m * m) / (s * s * s);

		if (num_box >= TPB) {
			grid.x = ceilf(num_box / (float)TPB); // m/s * m/s * m/s= (grid_size * TPB)
			grid.y = 1; grid.z = 1;
		}
		else {
			grid.x = 1; grid.y = 1; grid.z = 1;
			block.x = num_box; block.y = 1; block.z = 1;
			bits_TPB = log(num_box) / log(2);
		}

		// BCKernel call. Compute box-counting for grids of size s x s
		BCKernel3D << <grid, block >> > (device_M, m, bits_m, sm, ni + 1, bits_TPB, &device_n[ni]);

		ni++;
		s <<= 1;
		size >>= 1;
	}

	// GPU-CPU data transfer of the box-counting results
	cudaStatus = hipMemcpy(n, device_n, nn * sizeof(unsigned int), hipMemcpyDeviceToHost);

	hipFree(device_M);
	hipFree(device_n);
	return cudaStatus;
}

